#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
#include "common.h"
//#include <hip/hip_runtime.h>

// Structure for creature
struct Creature {
	int Energy; 
	int Velocity;
	int TimeLeft;
	int Code[100];
	int codelen, codepos;
	int ParentRef;
	int Ref;
	long Output[3];
	bool Child;
};

typedef struct Creature Creature;

// Structure for World
struct World {
	int Energy;
	long TimeLeft;
	struct Creature Lifes[32000];
	int NumOfLifes;
	int AliveCreatures;
	int MaxEnergy;
	int Input[3];
	long Fitness[3];
};

typedef struct World World;

// Return rnadom number between min and max 
int range_rand(int min_num, int max_num) {

	if(min_num > max_num) {
		fprintf(stderr, "min_num %i is greater than max_num %i!\n", min_num, max_num); 
	}
	// Return random number in range
	return min_num + (rand() % (max_num - min_num + 1));
} 

bool IsAlive(Creature *Life)
{
	if  (Life->Energy > 0 && Life->TimeLeft > 0) return(true);
	return(false);
}

Creature FindCreature(World *Iteration, int Ref)
{
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].Ref == Ref) return(Iteration->Lifes[i]);
	}
	return(Iteration->Lifes[0]);
}

void PrintCode(Creature *Life)
{
	for (int i = 0; i < Life->codelen; i++)
	printf("%i", Life->Code[i]);
}

// Calculate All World Energy  
int AllEnergy(World *Iteration)
{
	int totalenergy = 0;
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].TimeLeft > 0)
		totalenergy += Iteration->Lifes[i].Energy;
	}
	return(totalenergy);
}

void PrintLife(Creature *Life)
{
        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i OUTPUT:%ld#%ld#%ld# \nCode:",
        Life->Energy, Life->Velocity, Life->TimeLeft, Life->codelen, Life->codepos, Life->ParentRef, Life->Ref, Life->Output[0], Life->Output[1], Life->Output[2]);

        for (int k = 0; k < Life->codelen; k++) {
		if (k == Life->codepos) printf("*"); 
		printf("%i,", Life->Code[k]);
	}
}

Creature InitLife(World *Iteration, int ParRef)
{
	Creature Life;

	Life.Energy = Iteration->MaxEnergy - AllEnergy(Iteration); 
	if (Life.Energy > 5) Life.Energy = 15;

	Life.Velocity = 1;
	Life.TimeLeft = 19;
	Life.codelen = range_rand(5, 10);
	Life.codepos = 0;
	Life.Child = false;
	Life.Output[0] = 0;
	Life.Output[1] = 0;
	Life.Output[2] = 0;
	for (int i = 0; i < Life.codelen; i++) Life.Code[i] = range_rand(1, 9);
	Life.Ref = Iteration->NumOfLifes;
//	if (ParRef == 0) printf("\n *** REF IS BROKEN");
	Life.ParentRef = ParRef;

//	printf("\n LIFE BORN");
//	PrintLife(Life);

	Iteration->Lifes[Iteration->NumOfLifes] = Life;
	Iteration->NumOfLifes++;

	return(Life);
}

void RunLife(World *Iteration, const int n)
{
	struct Creature NewLife; // Make a child with random permutation

//	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

        Iteration->TimeLeft--;
//	Iteration->AliveCreatures = 0;
//	Iteration->Energy = 0;
//        printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i",
//        Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
	
	for (int i = 0; i < n; i++)
	{

	struct Creature Life = Iteration->Lifes[i];

	int NewRef = Life.Ref;

	// IsAlive
	if  (Life.Energy > 0 && Life.TimeLeft > 0)
	{
//		Iteration->Energy += Life.Energy;
//		Iteration->AliveCreatures++;

		// PrintLife	
//	        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
//	        Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
//		for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

//		Life.Output[0] = Life.Output[1] = Life.Output[2] = 0;
		Life.Output[0] = Iteration->Input[0];
                Life.Output[1] = Iteration->Input[1];
                Life.Output[2] = Iteration->Input[2];

		// run code "Velocity" number of times     
		for (int i = 0; i < Life.codelen; i++) {
		int k;
		switch(Life.Code[i])
		{
			case 1: Life.Energy += 2;
				break;
			case 2: Life.Velocity++; //if (Life.codelen > 3) Life.codelen = Life.codelen/2; // Half genome
				break;
			case 3: Life.Output[0] = Life.Output[0] * Life.Output[0];
				Life.Output[1] = Life.Output[1] * Life.Output[1];
				Life.Output[2] = Life.Output[2] * Life.Output[2];
				//for (k = 0; k < Life.codelen-1; k++) // Learn from myself? other creature
				//Life.Code[Life.codelen+k] = Life.Code[k+1];
				//Life.codelen = Life.codelen+k;
				break;
			case 4: //Life.Child = true;
				Life.Output[0]--;
				Life.Output[1]--;
				Life.Output[2]--;
				break;
			case 5: Life.Output[0]++;
				Life.Output[1]++;
				Life.Output[2]++;
				break;
			case 6: Life.Output[0] = Life.Output[0] + Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] + Iteration->Input[1];
				Life.Output[2] = Life.Output[2] + Iteration->Input[2];
				break;
                        case 7: Life.Output[0] = Life.Output[0] - Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] - Iteration->Input[1];
				Life.Output[2] = Life.Output[2] - Iteration->Input[2];
				break;
                        case 8: Life.Output[0] = Life.Output[0] * Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] * Iteration->Input[1];
				Life.Output[2] = Life.Output[2] * Iteration->Input[2];
				break;
                        case 9: Life.Output[0] = Life.Output[0] / Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] / Iteration->Input[1];
				Life.Output[2] = Life.Output[2] / Iteration->Input[2];
				break;
		}
		Life.codepos++;
		if (Life.codepos > Life.codelen) Life.codepos = 0;
		}
		Life.TimeLeft--;
		Life.Energy--;
	}
                // PrintLife
//                printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
//                Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
//                for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

		Iteration->Lifes[i] = Life;
	}
//        printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i",
//        Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

void NewWorld(World *Iteration)
{
        Iteration->Energy = 0;
        Iteration->TimeLeft = 1500000;
        Iteration->NumOfLifes = 0;
        Iteration->MaxEnergy = 50;
        Iteration->AliveCreatures = 0;
	int I0 = Iteration->Input[0] = 5;
//	Iteration->Fitness = ((((Iteration->Input + Iteration->Input + 1) * Iteration->Input) - Iteration->Input) / Iteration->Input) + Iteration->Input - 1;
	// Code:9,9,4,6,9,5,7,9,5,4,5,3,4,6,3,8,5,
//	Iteration->Fitness[0] = ()
	Iteration->Fitness[0] = (((I0 * I0) * I0 + 1 + I0) - 1) * I0;
        int I1 = Iteration->Input[1] = 10;
	Iteration->Fitness[1] = (((I1 * I1) * I1 + 1 + I1) - 1) * I1;
        int I2 = Iteration->Input[2] = 0;
        Iteration->Fitness[2] = (((I2 * I2) * I2 + 1 + I2) - 1) * I2;
	for (int i = 0; i < 2; i++)
	{
	        InitLife(Iteration, 0);
	}
	Creature ArtLife = InitLife(Iteration, -1);
//	ArtLife.Code = {5,1,8,2,6,6,1,3,3,1,6};
	ArtLife.Code[0] = 8;
	ArtLife.Code[1] = 4;
	ArtLife.codelen = 2;
}

void PrintWorld(World *Iteration)
{
	printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%ld Energy:%i NumOfLifes:%i AliveCreatures: %i\n--------------------", 
	Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

int main(int argc, char **argv)
{
        time_t t;

        // Intializes random number generator
        srand((unsigned) time(&t));

	// set up device
/*	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("device %d: %s \n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));	

	// allocate host memory
	int nElem = 1<22;*/
	size_t nBytes = sizeof(World);
	World     *h_A = (World *)malloc(nBytes);
	World *hostRef = (World *)malloc(nBytes);
	World *gpuRef  = (World *)malloc(nBytes);

	// initialize host array
	NewWorld(gpuRef);

	// allocate device memory
	World *d_A, *d_C;

//	CHECK(hipMalloc((World**)&d_A, nBytes));
//        CHECK(hipMalloc((World**)&d_C, nBytes));
	
	// copy data from host to device
//	CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));

        PrintLife(&gpuRef->Lifes[0]);
        PrintLife(&gpuRef->Lifes[1]);
        PrintLife(&h_A->Lifes[2]);

	PrintWorld(gpuRef);

        long BestFit[3];
	BestFit[0]  = abs(gpuRef->Fitness[0] - gpuRef->Lifes[0].Output[0]);
	BestFit[1]  = abs(gpuRef->Fitness[1] - gpuRef->Lifes[0].Output[1]);
	BestFit[2]  = abs(gpuRef->Fitness[2] - gpuRef->Lifes[0].Output[2]);
	
	int BestFitNo = 0;

        // Run World all iterations
	do
        {
                // copy data from host to device
//                CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));

//		RunLife <<<1, gpuRef->NumOfLifes>>>(d_A, 1<<22);
//	        RunLife <<<64, 512>>>(d_A, 1<<22);
		RunLife(gpuRef, gpuRef->NumOfLifes);
//		CHECK(hipDeviceSynchronize());
//	        CHECK(hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost));
		gpuRef->AliveCreatures = 0;
		gpuRef->Energy = 0;
	        BestFitNo = gpuRef->NumOfLifes-1;
		BestFit[0] = abs(gpuRef->Fitness[0] - gpuRef->Lifes[BestFitNo].Output[0]);
		BestFit[1] = abs(gpuRef->Fitness[1] - gpuRef->Lifes[BestFitNo].Output[1]);
	        BestFit[2] = abs(gpuRef->Fitness[2] - gpuRef->Lifes[BestFitNo].Output[2]);
		for (int j = 0; j < gpuRef->NumOfLifes; j++) {
//			PrintLife(&gpuRef->Lifes[j]);
//                        printf(">>%d", gpuRef->ChildLifes[j]);
			if (gpuRef->Lifes[j].Energy > 0 && gpuRef->Lifes[j].TimeLeft > 0) 
			{
//	                        PrintLife(&gpuRef->Lifes[j]);
				gpuRef->AliveCreatures++;
				gpuRef->Energy += gpuRef->Lifes[j].Energy;
//	                        PrintLife(&gpuRef->Lifes[j]);
//                                printf(" *** BestFit[0] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[0], gpuRef->Lifes[j].Output[0], abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]), BestFit[0]);
//                                printf(" *** BestFit[1] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[1], gpuRef->Lifes[j].Output[1], abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]), BestFit[1]);
//                                printf(" *** BestFit[2] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[2], gpuRef->Lifes[j].Output[2], abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]), BestFit[2]);
//			if (abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]) < BestFit[0] && abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]) < BestFit[1] && abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]) < BestFit[2]) {
			if (abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]) + abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]) + abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]) < BestFit[0] + BestFit[1] + BestFit[2]) {
				printf("\n *** BestFit vs NewBestFit : %ld# vs %ld#", BestFit[0] + BestFit[1] + BestFit[2], abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]) + abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]) + abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]));
				BestFit[0] = abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]);
                                BestFit[1] = abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]);
                                BestFit[2] = abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]);
				BestFitNo = j;
//				printf(" *** BestFit[0] = %ld - %ld = %ld", gpuRef->Lifes[j].Output[0], gpuRef->Fitness[0], BestFit[0]);
				if (BestFit[0] == 0 && BestFit[1] == 0 && BestFit[2] == 0) {
					PrintLife(&gpuRef->Lifes[j]);
	                                printf(" *** BestFit[0] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[0], gpuRef->Lifes[j].Output[0], abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]), BestFit[0]);
		                        printf(" *** BestFit[1] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[1], gpuRef->Lifes[j].Output[1], abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]), BestFit[1]);
			                printf(" *** BestFit[2] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[2], gpuRef->Lifes[j].Output[2], abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]), BestFit[2]);
					break;
				}
			}
			}
		}
		int p = 0;
		for (int n = 0; n < range_rand(10, 30); n++) 
		{
			for (p = p; p < gpuRef->NumOfLifes; p++) if (gpuRef->Lifes[p].TimeLeft <= 0 || gpuRef->Lifes[p].Energy <= 0) break;
			printf("\n ** Slot for new life is %i", p);
                        PrintLife(&gpuRef->Lifes[p]);
			gpuRef->Lifes[p].Energy = 29;
                        gpuRef->Lifes[p].TimeLeft = 29;
                        gpuRef->Lifes[p].Velocity = 1;
			if (range_rand(1, 4) == 1) {
				gpuRef->Lifes[p].codelen = gpuRef->Lifes[BestFitNo].codelen / 2;
			} else if (range_rand(1, 4) == 1) {
				gpuRef->Lifes[p].codelen = gpuRef->Lifes[BestFitNo].codelen * 2;
				if (gpuRef->Lifes[p].codelen > 49) gpuRef->Lifes[p].codelen = 49;
			} else {
				gpuRef->Lifes[p].codelen = gpuRef->Lifes[BestFitNo].codelen;
			}
                        gpuRef->Lifes[p].codepos = 0;
		        for (int k = 0; k < gpuRef->Lifes[BestFitNo].codelen; k++) {
				if (range_rand(1, 2) == 1) {
	                                gpuRef->Lifes[p].Code[k] = range_rand(1, 9);		
				}
				else { 
					gpuRef->Lifes[p].Code[k] = gpuRef->Lifes[BestFitNo].Code[k];
				}
			}
			gpuRef->Lifes[p].Ref = p;
                        gpuRef->Lifes[p].ParentRef = gpuRef->Lifes[BestFitNo].Ref;
			gpuRef->Lifes[p].Output[0] = 0;
                        gpuRef->Lifes[p].Output[1] = 0;
                        gpuRef->Lifes[p].Output[2] = 0;
                        PrintLife(&gpuRef->Lifes[BestFitNo]);
//			printf(" %ld#%ld#%ld#%ld", BestFit[0], BestFit[1], BestFit[2], BestFit[0] + BestFit[1] + BestFit[2]);
//                        printf("\n %ld#", BestFit[0] + BestFit[1] + BestFit[2]);

//				printf("\n *** Parent: %i", j);
//                        printf("\n ***LIFE IS BORN from %i", gpuRef->Lifes[BestFitNo].Ref);
                        PrintLife(&gpuRef->Lifes[p]);
                        if (p >= gpuRef->NumOfLifes) gpuRef->NumOfLifes++;
		}
                PrintWorld(gpuRef);
		// copy data from host to device
//	        CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));
		if (BestFit[0] == 0 && BestFit[1] == 0 && BestFit[2] == 0) break;
	} while (gpuRef->Energy > 0 && gpuRef->TimeLeft > 0);

//	CHECK(hipDeviceSynchronize());
//	CHECK(hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost));

	PrintWorld(gpuRef);

//	CHECK(hipGetLastError());;

	printf("\n\n ### THE WINNER IS %i", BestFitNo);
	PrintLife(&gpuRef->Lifes[BestFitNo]);

/*	printf("\n\n *** Admire the winners genomes history:");
        for (int i = 0; i < gpuRef->NumOfLifes; i++)
	{
		Creature Parent = gpuRef->Lifes[i];
		if (IsAlive(&Parent)) {
			PrintLife(&Parent);
			while (Parent.ParentRef > 0) {
				Parent = FindCreature(gpuRef, Parent.ParentRef);
				printf("->");
				PrintCode(&Parent);
//				PrintLife <<<1,1>>>(Parent);
			}
		}
	}*/
/*        printf("\n\n *** Admire the winners story:");
        for (int i = 0; i < gpuRef->NumOfLifes; i++)
        {
                Creature Parent = gpuRef->Lifes[i];
                if (IsAlive(&Parent)) {
//                        PrintLife(Parent);
			printf("\n");
                        while (Parent.ParentRef > 0) {
				printf("%i->", Parent.Ref);
                                Parent = FindCreature(gpuRef, Parent.ParentRef);
//                              PrintLife <<<1, 1>>>(Parent);
                        }
                }
        }
*/
	printf("\n");
}
