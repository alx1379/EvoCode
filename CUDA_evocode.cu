#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
#include "common.h"
//#include <hip/hip_runtime.h>

// Structure for creature
struct Creature {
	int Energy; 
	int Velocity;
	int TimeLeft;
	int Code[100];
	int codelen, codepos;
	int ParentRef;
	int Ref;
};

typedef struct Creature Creature;

// Structure for World
struct World {
	int Energy;
	int TimeLeft;
	struct Creature Lifes[5000];
	int NumOfLifes;
	int AliveCreatures;
	int MaxEnergy;
};

typedef struct World World;

// Return rnadom number between min and max 
int range_rand(int min_num, int max_num) {

	if(min_num > max_num) {
		fprintf(stderr, "min_num %i is greater than max_num %i!\n", min_num, max_num); 
	}
	// Return random number in range
	return min_num + (rand() % (max_num - min_num));
} 

bool IsAlive(Creature *Life)
{
	if  (Life->Energy > 0 && Life->TimeLeft > 0) return(true);
	return(false);
}

Creature FindCreature(World *Iteration, int Ref)
{
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].Ref == Ref) return(Iteration->Lifes[i]);
	}
}

void PrintCode(Creature *Life)
{
	for (int i = 0; i < Life->codelen; i++)
	printf("%i", Life->Code[i]);
}

// Calculate All World Energy  
int AllEnergy(World *Iteration)
{
	int totalenergy = 0;
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].TimeLeft > 0)
		totalenergy += Iteration->Lifes[i].Energy;
	}
	return(totalenergy);
}

void PrintLife(Creature *Life)
{
        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
        Life->Energy, Life->Velocity, Life->TimeLeft, Life->codelen, Life->codepos, Life->ParentRef, Life->Ref);

        for (int k = 0; k < Life->codelen; k++) printf("%i", Life->Code[k]);
}

Creature InitLife(World *Iteration, int ParRef)
{
	Creature Life;

	Life.Energy = Iteration->MaxEnergy - AllEnergy(Iteration); 
	if (Life.Energy > 5) Life.Energy = 5;

	Life.Velocity = 1;
	Life.TimeLeft = 5;
	Life.codelen = range_rand(5, 10);
	Life.codepos = 0;
	for (int i = 0; i < Life.codelen; i++) Life.Code[i] = range_rand(1, 5);
//	Life.Ref = range_rand(1, 65535);
	Life.Ref = Iteration->NumOfLifes;
	if (ParRef == 0) printf("\n *** REF IS BROKEN");
	Life.ParentRef = ParRef;

//	printf("\n LIFE BORN");
//	PrintLife(Life);

	Iteration->Lifes[Iteration->NumOfLifes] = Life;
	Iteration->NumOfLifes++;

	return(Life);
}

__global__ void RunLife(World *Iteration, const int n)
{
//	struct Creature New; // Make a child with random permutation

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < n)
	{

	struct Creature Life = Iteration->Lifes[i];

	int NewRef = Life.Ref;

	// IsAlive
	if  (Life.Energy > 0 && Life.TimeLeft > 0)
	{
		// PrintLife	
	        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
	        Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
		for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

		Iteration->AliveCreatures++;

		// run code "Velocity" number of times     
		for (int i = 0; i < Life.Velocity; i++) {
		int k;
		switch(Life.Code[Life.codepos])
		{
			case 1: Life.Energy += 2; // Feed
				break;
			case 2: if (Life.codelen > 3) Life.codelen = Life.codelen/2; // Half genome
				break;
			case 3: 
				for (k = 0; k < Life.codelen-1; k++) // Learn from other creature
				Life.Code[Life.codelen+k] = Life.Code[k+1];
				Life.codelen = Life.codelen+k;
				break;
/*			case 4: New = InitLife(Iteration, Life.Ref);
				if (New.codelen < 9 && range_rand(1, 3) == 1) { // 1/3 likelyhood of permutation for short genome
					New.Code[New.codelen] = range_rand(1, 5); // add new code at the end
					New.codelen++;
				} else {
					New.Code[range_rand(1, New.codelen-1)] = range_rand(1, 5); // 100% likelyhood of permutation for long genome and short that out of 1/3
				}
				NewRef = New.Ref;
				break;*/
			case 5: Life.Velocity++;
				break;
		}
		Life.codepos++;
		}
		Life.TimeLeft--;
		Life.Energy--;
	}
                // PrintLife
//                printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
//                Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
//                for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

		Iteration->Lifes[i] = Life;
	}

//	return(NewRef);
}

World InitWorld(void)
{
        World Iteration;

        Iteration.Energy = 0;
        Iteration.TimeLeft = 100;
	Iteration.NumOfLifes = 0;
	Iteration.MaxEnergy = 50;
	Iteration.AliveCreatures = 0;
	InitLife(&Iteration, 0);
	InitLife(&Iteration, 0);

	return(Iteration);
}

void NewWorld(World *Iteration)
{
        Iteration->Energy = 0;
        Iteration->TimeLeft = 100;
        Iteration->NumOfLifes = 0;
        Iteration->MaxEnergy = 50;
        Iteration->AliveCreatures = 0;
        InitLife(Iteration, 0);
        InitLife(Iteration, 0);
}

void PrintWorld(World *Iteration)
{
	printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i", 
	Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

// Run World Iteration
void RunWorld(World *Iteration)
{
	Iteration->Energy = AllEnergy(Iteration);
	Iteration->TimeLeft--;

	PrintWorld(Iteration);

//	int i;
//	scanf("%i", &i);

	Iteration->AliveCreatures = 0;

	RunLife <<<1, 10>>>(Iteration, 1<<22);
	hipDeviceSynchronize();

//	for (int i = 0; i < Iteration->NumOfLifes; i++) 
//	{
//		printf("\n Life number: %i", i);
//		int CurRef = RunLife <<<1, 1>>>(Iteration, &Iteration->Lifes[i]);
//		RunLife <<<1, 1>>>(Iteration, &Iteration->Lifes[i]);
//	}

	if (Iteration->TimeLeft > 0 && Iteration->Energy > 0) RunWorld(Iteration);
}

__global__ void helloFromGPU(void)
{
  printf("Hello World from GPU thread");
}

int main(int argc, char **argv)
{
        time_t t;

        // Intializes random number generator
        srand((unsigned) time(&t));

//	World NewWorld = InitWorld();
//        NewWorld.AliveCreatures = 0;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s test struct of array at ", argv[0]);
    printf("device %d: %s \n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));	

    // allocate host memory
    int nElem = 1<22;
    size_t nBytes = sizeof(World);
    World     *h_A = (World *)malloc(nBytes);
    World *hostRef = (World *)malloc(nBytes);
    World *gpuRef  = (World *)malloc(nBytes);

	// initialize host array
	NewWorld(h_A);

   // allocate device memory
    World *d_A, *d_C;
    CHECK(hipMalloc((World**)&d_A, nBytes));
    CHECK(hipMalloc((World**)&d_C, nBytes));

   // copy data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

        RunLife <<<1, 10>>>(d_A, 1<<22);
	CHECK(hipDeviceSynchronize());
	CHECK(hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost));

	PrintLife(&gpuRef->Lifes[0]);
        PrintLife(&gpuRef->Lifes[1]);

    CHECK(hipGetLastError());


//	RunWorld(&NewWorld);

	printf("\n\n *** Admire the winners genomes history:");
/*        for (int i = 0; i < NewWorld.NumOfLifes; i++)
	{
		Creature Parent = NewWorld.Lifes[i];
		if (IsAlive(&Parent)) {
			PrintLife(&Parent);
			while (Parent.ParentRef > 0) {
				Parent = FindCreature(&NewWorld, Parent.ParentRef);
				printf("->");
				PrintCode(&Parent);
//				PrintLife <<<1,1>>>(Parent);
			}
		}
	}
        printf("\n\n *** Admire the winners story:");
        for (int i = 0; i < NewWorld.NumOfLifes; i++)
        {
                Creature Parent = NewWorld.Lifes[i];
                if (IsAlive(&Parent)) {
//                        PrintLife(Parent);
			printf("\n");
                        while (Parent.ParentRef > 0) {
				printf("%i->", Parent.Ref);
                                Parent = FindCreature(&NewWorld, Parent.ParentRef);
//                              PrintLife <<<1, 1>>>(Parent);
                        }
                }
        }*/


	printf("\n");
}
