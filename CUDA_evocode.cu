
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
//#include <cuda_runtime.h>

// Structure for creature
struct Creature {
	int Energy; 
	int Velocity;
	int TimeLeft;
	int Code[100];
	int codelen, codepos;
	int ParentRef;
	int Ref;
};

typedef struct Creature Creature;

// Structure for World
struct World {
	int Energy;
	int TimeLeft;
	struct Creature Lifes[5000];
	int NumOfLifes;
	int AliveCreatures;
	int MaxEnergy;
};

typedef struct World World;

// Return rnadom number between min and max 
int range_rand(int min_num, int max_num) {

	if(min_num > max_num) {
		fprintf(stderr, "min_num %i is greater than max_num %i!\n", min_num, max_num); 
	}
	// Return random number in range
	return min_num + (rand() % (max_num - min_num));
} 

bool IsAlive(Creature *Life)
{
	if  (Life->Energy > 0 && Life->TimeLeft > 0) return(true);
	return(false);
}

Creature FindCreature(World *Iteration, int Ref)
{
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].Ref == Ref) return(Iteration->Lifes[i]);
	}
}

void PrintCode(Creature *Life)
{
	for (int i = 0; i < Life->codelen; i++)
	printf("%i", Life->Code[i]);
}

// Calculate All World Energy  
int AllEnergy(World *Iteration)
{
	int totalenergy = 0;
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].TimeLeft > 0)
		totalenergy += Iteration->Lifes[i].Energy;
	}
	return(totalenergy);
}

void PrintLife(Creature *Life)
{
        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
        Life->Energy, Life->Velocity, Life->TimeLeft, Life->codelen, Life->codepos, Life->ParentRef, Life->Ref);

        for (int k = 0; k < Life->codelen; k++) printf("%i", Life->Code[k]);
}

Creature InitLife(World *Iteration, int ParRef)
{
	Creature Life;

	Life.Energy = Iteration->MaxEnergy - AllEnergy(Iteration); 
	if (Life.Energy > 5) Life.Energy = 5;

	Life.Velocity = 1;
	Life.TimeLeft = 5;
	Life.codelen = range_rand(5, 10);
	Life.codepos = 0;
	for (int i = 0; i < Life.codelen; i++) Life.Code[i] = range_rand(1, 5);
//	Life.Ref = range_rand(1, 65535);
	Life.Ref = Iteration->NumOfLifes;
	if (ParRef == 0) printf("\n *** REF IS BROKEN");
	Life.ParentRef = ParRef;

//	printf("\n LIFE BORN");
//	PrintLife(Life);

	Iteration->Lifes[Iteration->NumOfLifes] = Life;
	Iteration->NumOfLifes++;

	return(Life);
}

int RunLife(World *Iteration, Creature *Life)
{
	struct Creature New; // Make a child with random permutation

	int NewRef = Life->Ref;

	if (IsAlive(Life))
	{
		PrintLife(Life);
		Iteration->AliveCreatures++;

		// run code "Velocity" number of times     
		for (int i = 0; i < Life->Velocity; i++) {
		int k;
		switch(Life->Code[Life->codepos])
		{
			case 1: Life->Energy += 2; // Feed
				break;
			case 2: if (Life->codelen > 3) Life->codelen -= range_rand(1, Life->codelen/2); // Half genome
				break;
			case 3: 
				for (k = 0; k < Life->codelen-1; k++) // Learn from other creature
				Life->Code[Life->codelen+k] = Life->Code[k+1];
				Life->codelen = Life->codelen+k;
				break;
			case 4: New = InitLife(Iteration, Life->Ref);
				if (New.codelen < 9 && range_rand(1, 3) == 1) { // 1/3 likelyhood of permutation for short genome
					New.Code[New.codelen] = range_rand(1, 5); // add new code at the end
					New.codelen++;
				} else {
					New.Code[range_rand(1, New.codelen-1)] = range_rand(1, 5); // 100% likelyhood of permutation for long genome and short that out of 1/3
				}
				NewRef = New.Ref;
				break;
			case 5: Life->Velocity++;
				break;
		}
		Life->codepos++;
		}
		Life->TimeLeft--;
		Life->Energy--;
	}
	return(NewRef);
}

World InitWorld(void)
{
        World Iteration;

        Iteration.Energy = 0;
        Iteration.TimeLeft = 100;
	Iteration.NumOfLifes = 0;
	Iteration.MaxEnergy = 50;
	Iteration.AliveCreatures = 0;
	InitLife(&Iteration, 0);
	InitLife(&Iteration, 0);

	return(Iteration);
}

void PrintWorld(World *Iteration)
{
	printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i", 
	Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

// Run World Iteration
void RunWorld(World *Iteration)
{
	Iteration->Energy = AllEnergy(Iteration);
	Iteration->TimeLeft--;

	PrintWorld(Iteration);

//	int i;
//	scanf("%i", &i);

	Iteration->AliveCreatures = 0;
	for (int i = 0; i < Iteration->NumOfLifes; i++) 
	{
//		printf("\n Life number: %i", i);
		int CurRef = RunLife(Iteration, &Iteration->Lifes[i]);
	}

	if (Iteration->TimeLeft > 0 && Iteration->Energy > 0) RunWorld(Iteration);
}

__global__ void helloFromGPU(void)
{
  printf("Hello World from GPU thread");
}

int main(void)
{
        time_t t;

        // Intializes random number generator
        srand((unsigned) time(&t));

	helloFromGPU <<<1, 512>>>();
	hipDeviceSynchronize();
	//  cudaDeviceReset();

	World NewWorld = InitWorld();
//	RunWorld <<<1, 10>>>(&NewWorld);
	RunWorld(&NewWorld);

	printf("\n\n *** Admire the winners genomes history:");
        for (int i = 0; i < NewWorld.NumOfLifes; i++)
	{
		Creature Parent = NewWorld.Lifes[i];
		if (IsAlive(&Parent)) {
			PrintLife(&Parent);
			while (Parent.ParentRef > 0) {
				Parent = FindCreature(&NewWorld, Parent.ParentRef);
				printf("->");
				PrintCode(&Parent);
//				PrintLife(Parent);
			}
		}
	}
        printf("\n\n *** Admire the winners story:");
        for (int i = 0; i < NewWorld.NumOfLifes; i++)
        {
                Creature Parent = NewWorld.Lifes[i];
                if (IsAlive(&Parent)) {
//                        PrintLife(Parent);
			printf("\n");
                        while (Parent.ParentRef > 0) {
				printf("%i->", Parent.Ref);
                                Parent = FindCreature(&NewWorld, Parent.ParentRef);
//                              PrintLife(Parent);
                        }
                }
        }


	printf("\n");
}
