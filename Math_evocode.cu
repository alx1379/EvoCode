#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
#include "common.h"
//#include <hip/hip_runtime.h>

// Structure for creature
struct Creature {
	int Energy; 
	int Velocity;
	int TimeLeft;
	int Code[100];
	int codelen, codepos;
	int ParentRef;
	int Ref;
	long Output[3];
	bool Child;
};

typedef struct Creature Creature;

// Structure for World
struct World {
	int Energy;
	int TimeLeft;
	struct Creature Lifes[50000];
//	bool ChildLifes[5000];
	int NumOfLifes;
	int AliveCreatures;
	int MaxEnergy;
	int Input[3];
	int Fitness[3];
};

typedef struct World World;

// Return rnadom number between min and max 
int range_rand(int min_num, int max_num) {

	if(min_num > max_num) {
		fprintf(stderr, "min_num %i is greater than max_num %i!\n", min_num, max_num); 
	}
	// Return random number in range
	return min_num + (rand() % (max_num - min_num + 1));
} 

bool IsAlive(Creature *Life)
{
	if  (Life->Energy > 0 && Life->TimeLeft > 0) return(true);
	return(false);
}

Creature FindCreature(World *Iteration, int Ref)
{
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].Ref == Ref) return(Iteration->Lifes[i]);
	}
}

void PrintCode(Creature *Life)
{
	for (int i = 0; i < Life->codelen; i++)
	printf("%i", Life->Code[i]);
}

// Calculate All World Energy  
int AllEnergy(World *Iteration)
{
	int totalenergy = 0;
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].TimeLeft > 0)
		totalenergy += Iteration->Lifes[i].Energy;
	}
	return(totalenergy);
}

void PrintLife(Creature *Life)
{
        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i OUTPUT:%i#%i#%i# \nCode:",
        Life->Energy, Life->Velocity, Life->TimeLeft, Life->codelen, Life->codepos, Life->ParentRef, Life->Ref, Life->Output[0], Life->Output[1], Life->Output[2]);

        for (int k = 0; k < Life->codelen; k++) {
		if (k == Life->codepos) printf("*"); 
		printf("%i,", Life->Code[k]);
	}
}

Creature InitLife(World *Iteration, int ParRef)
{
	Creature Life;

	Life.Energy = Iteration->MaxEnergy - AllEnergy(Iteration); 
	if (Life.Energy > 5) Life.Energy = 15;

	Life.Velocity = 1;
	Life.TimeLeft = 19;
	Life.codelen = range_rand(5, 10);
	Life.codepos = 0;
	Life.Child = false;
	Life.Output[0] = 0;
	Life.Output[1] = 0;
	Life.Output[2] = 0;
	for (int i = 0; i < Life.codelen; i++) Life.Code[i] = range_rand(1, 9);
//	Life.Ref = range_rand(1, 65535);
	Life.Ref = Iteration->NumOfLifes;
//	if (ParRef == 0) printf("\n *** REF IS BROKEN");
	Life.ParentRef = ParRef;

//	printf("\n LIFE BORN");
//	PrintLife(Life);

	Iteration->Lifes[Iteration->NumOfLifes] = Life;
	Iteration->NumOfLifes++;

	return(Life);
}

void NewLife(World *Iteration, int ParRef, Creature *Life)
{
        Life->Energy = Iteration->MaxEnergy - Iteration->Energy;
        if (Life->Energy > 5) Life->Energy = 15;

        Life->Velocity = 1;
        Life->TimeLeft = 19;
        Life->codelen = range_rand(5, 10);
        Life->codepos = 0;
	Life->Child = false;
	Life->Output[0] = 0;
	Life->Output[1] = 0;
	Life->Output[2] = 0;
        for (int i = 0; i < Life->codelen; i++) Life->Code[i] = range_rand(1, 9);
        Life->Ref = Iteration->NumOfLifes;
//        if (ParRef == 0) printf("\n *** REF IS BROKEN");
        Life->ParentRef = ParRef;

//      printf("\n LIFE BORN");
//      PrintLife(Life);

        Iteration->Lifes[Iteration->NumOfLifes] = *Life;
        Iteration->NumOfLifes++;
}

__global__ void RunLife(World *Iteration, const int n)
{
	struct Creature NewLife; // Make a child with random permutation

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

        Iteration->TimeLeft--;
//	Iteration->AliveCreatures = 0;
//	Iteration->Energy = 0;
//        printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i",
//        Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
	
	if (i < n)
	{

	struct Creature Life = Iteration->Lifes[i];

	int NewRef = Life.Ref;

	// IsAlive
	if  (Life.Energy > 0 && Life.TimeLeft > 0)
	{
//		Iteration->Energy += Life.Energy;
//		Iteration->AliveCreatures++;

		// PrintLife	
//	        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
//	        Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
//		for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

		// run code "Velocity" number of times     
		for (int i = 0; i < Life.Velocity; i++) {
		int k;
		switch(Life.Code[Life.codepos])
		{
			case 1: Life.Energy += 2;
				break;
			case 2: Life.Velocity++; //if (Life.codelen > 3) Life.codelen = Life.codelen/2; // Half genome
				break;
			case 3: Life.Output[0] = Life.Output[0] * Life.Output[0];
				Life.Output[1] = Life.Output[1] * Life.Output[1];
				Life.Output[2] = Life.Output[2] * Life.Output[2];
				//for (k = 0; k < Life.codelen-1; k++) // Learn from myself? other creature
				//Life.Code[Life.codelen+k] = Life.Code[k+1];
				//Life.codelen = Life.codelen+k;
				break;
			case 4: //Life.Child = true;
				Life.Output[0]--;
				Life.Output[1]--;
				Life.Output[2]--;
				break;
			case 5: Life.Output[0]++;
				Life.Output[1]++;
				Life.Output[2]++;
				break;
			case 6: Life.Output[0] = Life.Output[0] + Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] + Iteration->Input[1];
				Life.Output[2] = Life.Output[2] + Iteration->Input[2];
				break;
                        case 7: Life.Output[0] = Life.Output[0] - Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] - Iteration->Input[1];
				Life.Output[2] = Life.Output[2] - Iteration->Input[2];
				break;
                        case 8: Life.Output[0] = Life.Output[0] * Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] * Iteration->Input[1];
				Life.Output[2] = Life.Output[2] * Iteration->Input[2];
				break;
                        case 9: Life.Output[0] = Life.Output[0] / Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] / Iteration->Input[1];
				Life.Output[2] = Life.Output[2] / Iteration->Input[2];
				break;
		}
		Life.codepos++;
		if (Life.codepos > Life.codelen) Life.codepos = 0;
		}
		Life.TimeLeft--;
		Life.Energy--;
	}
                // PrintLife
//                printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
//                Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
//                for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

		Iteration->Lifes[i] = Life;
	}
//        printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i",
//        Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

/*World InitWorld(void)
{
        World Iteration;

        Iteration.Energy = 0;
        Iteration.TimeLeft = 200;
	Iteration.NumOfLifes = 0;
	Iteration.MaxEnergy = 50;
	Iteration.AliveCreatures = 0;
	Iteration.Input = 0;
        Iteration.Fitness = ((((Iteration.Input + Iteration.Input) * Iteration.Input) - Iteration.Input) / Iteration.Input) + Iteration.Input;
	InitLife(&Iteration, 0);
	InitLife(&Iteration, 0);

	return(Iteration);
}*/

void NewWorld(World *Iteration)
{
        Iteration->Energy = 0;
        Iteration->TimeLeft = 20000;
        Iteration->NumOfLifes = 0;
        Iteration->MaxEnergy = 50;
        Iteration->AliveCreatures = 0;
	Iteration->Input[0] = 5;
//	Iteration->Fitness = ((((Iteration->Input + Iteration->Input + 1) * Iteration->Input) - Iteration->Input) / Iteration->Input) + Iteration->Input - 1;
	Iteration->Fitness[0] = (Iteration->Input[0] * Iteration->Input[0]) * Iteration->Input[0] + 1;
//	Iteration->Fitness[0] = 1;
        Iteration->Input[1] = 10;
	Iteration->Fitness[1] = (Iteration->Input[1] * Iteration->Input[1]) * Iteration->Input[1] + 1;
//	Iteration->Fitness[1] = 1;
        Iteration->Input[2] = 0;
        Iteration->Fitness[2] = (Iteration->Input[2] * Iteration->Input[2]) * Iteration->Input[2] + 1;
//        Iteration->Fitness[2] = 1;
	for (int i = 0; i < 2; i++)
	{
	        InitLife(Iteration, 0);
	}
	Creature ArtLife = InitLife(Iteration, -1);
//	ArtLife.Code = {5,1,8,2,6,6,1,3,3,1,6};
	ArtLife.Code[0] = 8;
	ArtLife.Code[1] = 4;
	ArtLife.codelen = 2;
}

void PrintWorld(World *Iteration)
{
	printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i\n--------------------", 
	Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

// Run World Iteration
void RunWorld(World *Iteration)
{
	Iteration->Energy = AllEnergy(Iteration);
	Iteration->TimeLeft--;

	PrintWorld(Iteration);

//	int i;
//	scanf("%i", &i);

	Iteration->AliveCreatures = 0;

	RunLife <<<1, 10>>>(Iteration, 1<<22);
	hipDeviceSynchronize();

//	for (int i = 0; i < Iteration->NumOfLifes; i++) 
//	{
//		printf("\n Life number: %i", i);
//		int CurRef = RunLife <<<1, 1>>>(Iteration, &Iteration->Lifes[i]);
//		RunLife <<<1, 1>>>(Iteration, &Iteration->Lifes[i]);
//	}

	if (Iteration->TimeLeft > 0 && Iteration->Energy > 0) RunWorld(Iteration);
}

__global__ void helloFromGPU(void)
{
  printf("Hello World from GPU thread");
}

int main(int argc, char **argv)
{
        time_t t;

        // Intializes random number generator
        srand((unsigned) time(&t));

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("%s test struct of array at ", argv[0]);
	printf("device %d: %s \n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));	

	// allocate host memory
	int nElem = 1<22;
	size_t nBytes = sizeof(World);
	World     *h_A = (World *)malloc(nBytes);
	World *hostRef = (World *)malloc(nBytes);
	World *gpuRef  = (World *)malloc(nBytes);

	// initialize host array
	NewWorld(gpuRef);

	// allocate device memory
	World *d_A, *d_C;

	CHECK(hipMalloc((World**)&d_A, nBytes));
        CHECK(hipMalloc((World**)&d_C, nBytes));
	
	// copy data from host to device
//	CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));

        PrintLife(&gpuRef->Lifes[0]);
        PrintLife(&gpuRef->Lifes[1]);
        PrintLife(&h_A->Lifes[2]);

	PrintWorld(gpuRef);

        int BestFit[3];
	BestFit[0]  = abs(gpuRef->Fitness[0] - gpuRef->Lifes[0].Output[0]);
	BestFit[1]  = abs(gpuRef->Fitness[1] - gpuRef->Lifes[0].Output[1]);
	BestFit[2]  = abs(gpuRef->Fitness[2] - gpuRef->Lifes[0].Output[2]);

        // Run World all iterations
//        for (int i = 0; i < 200; i++)
	do
        {
//                for (int j = 0; j < gpuRef->NumOfLifes; j++) {
//			gpuRef->ChildLifes[j] = false;
//			printf(">>%d", gpuRef->ChildLifes[j]);
//		}

                // copy data from host to device
                CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));

//		RunLife <<<1, gpuRef->NumOfLifes>>>(d_A, 1<<22);
	        RunLife <<<1, 512>>>(d_A, 1<<22);
		CHECK(hipDeviceSynchronize());
	        CHECK(hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost));
		gpuRef->AliveCreatures = 0;
		gpuRef->Energy = 0;
		BestFit[0] = abs(gpuRef->Fitness[0] - gpuRef->Lifes[0].Output[0]);
		BestFit[1] = abs(gpuRef->Fitness[1] - gpuRef->Lifes[0].Output[1]);
	        BestFit[2] = abs(gpuRef->Fitness[2] - gpuRef->Lifes[0].Output[2]);
		int BestFitNo = 0;
		for (int j = 0; j < gpuRef->NumOfLifes; j++) {
//			PrintLife(&gpuRef->Lifes[j]);
//                        printf(">>%d", gpuRef->ChildLifes[j]);
			if (gpuRef->Lifes[j].Energy > 0 && gpuRef->Lifes[j].TimeLeft > 0) {
//	                        PrintLife(&gpuRef->Lifes[j]);
				gpuRef->AliveCreatures++;
				gpuRef->Energy += gpuRef->Lifes[j].Energy;
//	                        PrintLife(&gpuRef->Lifes[j]);
//                                printf(" *** BestFit[0] = %i - %i = %i vs CurBestFit %i", gpuRef->Fitness[0], gpuRef->Lifes[j].Output[0], abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]), BestFit[0]);
//                                printf(" *** BestFit[1] = %i - %i = %i vs CurBestFit %i", gpuRef->Fitness[1], gpuRef->Lifes[j].Output[1], abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]), BestFit[1]);
//                                printf(" *** BestFit[2] = %i - %i = %i vs CurBestFit %i", gpuRef->Fitness[2], gpuRef->Lifes[j].Output[2], abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]), BestFit[2]);
			if (abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]) < BestFit[0] && abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]) < BestFit[1] && abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]) < BestFit[2]) {
				BestFit[0] = abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]);
                                BestFit[1] = abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]);
                                BestFit[2] = abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]);
				BestFitNo = j;
//				printf(" *** BestFit[0] = %i - %i = %i", gpuRef->Lifes[j].Output[0], gpuRef->Fitness[0], BestFit[0]);
				if (BestFit[0] == 0 && BestFit[1] == 0 && BestFit[2] == 0) {
					PrintLife(&gpuRef->Lifes[j]);
	                                printf(" *** BestFit[0] = %i - %i = %i vs CurBestFit %i", gpuRef->Fitness[0], gpuRef->Lifes[j].Output[0], abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]), BestFit[0]);
		                        printf(" *** BestFit[1] = %i - %i = %i vs CurBestFit %i", gpuRef->Fitness[1], gpuRef->Lifes[j].Output[1], abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]), BestFit[1]);
			                printf(" *** BestFit[2] = %i - %i = %i vs CurBestFit %i", gpuRef->Fitness[2], gpuRef->Lifes[j].Output[2], abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]), BestFit[2]);

					break;
				}
			}
			}
		}
//			if (gpuRef->Lifes[j].Child == true) 
//			{
//				gpuRef->Lifes[j].Child = false;
//				printf("\n ***LIFE IS BORN from %i", gpuRef->Lifes[BestFitNo].Ref);
//                                PrintLife(&gpuRef->Lifes[BestFitNo]);
			gpuRef->Lifes[gpuRef->NumOfLifes].Energy = 15;
                        gpuRef->Lifes[gpuRef->NumOfLifes].TimeLeft = 19;
                        gpuRef->Lifes[gpuRef->NumOfLifes].Velocity = 1;
                        gpuRef->Lifes[gpuRef->NumOfLifes].codelen = gpuRef->Lifes[BestFitNo].codelen;
                        gpuRef->Lifes[gpuRef->NumOfLifes].codepos = 0;
		        for (int k = 0; k < gpuRef->Lifes[BestFitNo].codelen; k++) {
				if (range_rand(1, 3) == 1) {
	                                gpuRef->Lifes[gpuRef->NumOfLifes].Code[k] = range_rand(1, 9);		
				}
				else { 
					gpuRef->Lifes[gpuRef->NumOfLifes].Code[k] = gpuRef->Lifes[BestFitNo].Code[k];
				}
			}
			gpuRef->Lifes[gpuRef->NumOfLifes].Ref = gpuRef->NumOfLifes;
                        gpuRef->Lifes[gpuRef->NumOfLifes].ParentRef = gpuRef->Lifes[BestFitNo].Ref;
//				printf("\n *** Parent: %i", j);
//                        printf("\n ***LIFE IS BORN from %i", gpuRef->Lifes[BestFitNo].Ref);
//                        PrintLife(&gpuRef->Lifes[gpuRef->NumOfLifes]);
                        gpuRef->NumOfLifes++;
//			}
//		}
                PrintWorld(gpuRef);
		// copy data from host to device
//	        CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));
		if (BestFit[0] == 0 && BestFit[1] == 0 && BestFit[2] == 0) break;
	} while (gpuRef->Energy > 0 && gpuRef->TimeLeft > 0);

	CHECK(hipDeviceSynchronize());
	CHECK(hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost));

	PrintWorld(gpuRef);

//	PrintLife(&gpuRef->Lifes[0]);
//        PrintLife(&gpuRef->Lifes[1]);
//        PrintLife(&gpuRef->Lifes[2]);

	CHECK(hipGetLastError());

//	RunWorld(&NewWorld);

	printf("\n\n *** Admire the winners genomes history:");
        for (int i = 0; i < gpuRef->NumOfLifes; i++)
	{
		Creature Parent = gpuRef->Lifes[i];
		if (IsAlive(&Parent)) {
			PrintLife(&Parent);
			while (Parent.ParentRef > 0) {
				Parent = FindCreature(gpuRef, Parent.ParentRef);
				printf("->");
				PrintCode(&Parent);
//				PrintLife <<<1,1>>>(Parent);
			}
		}
	}
        printf("\n\n *** Admire the winners story:");
        for (int i = 0; i < gpuRef->NumOfLifes; i++)
        {
                Creature Parent = gpuRef->Lifes[i];
                if (IsAlive(&Parent)) {
//                        PrintLife(Parent);
			printf("\n");
                        while (Parent.ParentRef > 0) {
				printf("%i->", Parent.Ref);
                                Parent = FindCreature(gpuRef, Parent.ParentRef);
//                              PrintLife <<<1, 1>>>(Parent);
                        }
                }
        }

	printf("\n");
}
