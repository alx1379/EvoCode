#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdbool.h>
#include "common.h"
#include <math.h>
//#include <hip/hip_runtime.h>

// Structure for creature
struct Creature {
	int Energy; 
	int Velocity;
	int TimeLeft;
	int Code[100];
	int codelen, codepos;
	int ParentRef;
	int Ref;
	char Output[3][50];
	bool Child;
};

typedef struct Creature Creature;

// Structure for World
struct World {
	int Energy;
	long TimeLeft;
	struct Creature Lifes[32000];
	int NumOfLifes;
	int AliveCreatures;
	int MaxEnergy;
	char Input[3][50];
	char Fitness[3][50];
};

typedef struct World World;

// Return rnadom number between min and max 
int range_rand(int min_num, int max_num) {

	if(min_num > max_num) {
		fprintf(stderr, "min_num %i is greater than max_num %i!\n", min_num, max_num); 
	}
	// Return random number in range
	return min_num + (rand() % (max_num - min_num + 1));
} 

bool IsAlive(Creature *Life)
{
	if  (Life->Energy > 0 && Life->TimeLeft > 0) return(true);
	return(false);
}

Creature FindCreature(World *Iteration, int Ref)
{
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].Ref == Ref) return(Iteration->Lifes[i]);
	}
	return(Iteration->Lifes[0]);
}

void PrintCode(Creature *Life)
{
	for (int i = 0; i < Life->codelen; i++)
	printf("%i", Life->Code[i]);
}

// Calculate All World Energy  
int AllEnergy(World *Iteration)
{
	int totalenergy = 0;
	for (int i = 0; i < Iteration->NumOfLifes; i++)
	{
		if (Iteration->Lifes[i].TimeLeft > 0)
		totalenergy += Iteration->Lifes[i].Energy;
	}
	return(totalenergy);
}

void PrintLife(Creature *Life)
{
        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i OUTPUT:%s#%s#%s# \nCode:",
        Life->Energy, Life->Velocity, Life->TimeLeft, Life->codelen, Life->codepos, Life->ParentRef, Life->Ref, Life->Output[0], Life->Output[1], Life->Output[2]);

        for (int k = 0; k < Life->codelen; k++) {
		if (k == Life->codepos) printf("*"); 
		printf("%i,", Life->Code[k]);
	}
}

Creature InitLife(World *Iteration, int ParRef)
{
	Creature Life;

	Life.Energy = Iteration->MaxEnergy - AllEnergy(Iteration); 
	if (Life.Energy > 5) Life.Energy = 15;

	Life.Velocity = 1;
	Life.TimeLeft = 19;
	Life.codelen = range_rand(5, 10);
	Life.codepos = 0;
	Life.Child = false;
	strcpy(Life.Output[0], "0");
	strcpy(Life.Output[1], "0");
	strcpy(Life.Output[2], "0");
	for (int i = 0; i < Life.codelen; i++) Life.Code[i] = range_rand(1, 9);
	Life.Ref = Iteration->NumOfLifes;
//	if (ParRef == 0) printf("\n *** REF IS BROKEN");
	Life.ParentRef = ParRef;

//	printf("\n LIFE BORN");
//	PrintLife(Life);

	Iteration->Lifes[Iteration->NumOfLifes] = Life;
	Iteration->NumOfLifes++;

	return(Life);
}

__global__ void RunLife(World *Iteration, const int n)
{
	struct Creature NewLife; // Make a child with random permutation

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

        Iteration->TimeLeft--;
//	Iteration->AliveCreatures = 0;
//	Iteration->Energy = 0;
//        printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i",
//        Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
	
	if (i < n)
	{

	struct Creature Life = Iteration->Lifes[i];

	int NewRef = Life.Ref;

	// IsAlive
	if  (Life.Energy > 0 && Life.TimeLeft > 0)
	{
//		Iteration->Energy += Life.Energy;
//		Iteration->AliveCreatures++;

		// PrintLife	
//	        printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
//	        Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
//		for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

//		Life.Output[0] = Life.Output[1] = Life.Output[2] = "";
//		strcpy(Life.Output[0], Iteration->Input[0]);
//                strcpy(Life.Output[1], Iteration->Input[1]);
//	        strcpy(Life.Output[2], Iteration->Input[2]);

		// run code "Velocity" number of times     
		for (int i = 0; i < Life.codelen; i++) {
		int k;
		switch(Life.Code[i])
		{
			case 1: Life.Energy += 2;
				break;
			case 2: Life.Velocity++; //if (Life.codelen > 3) Life.codelen = Life.codelen/2; // Half genome
				break;
			case 3: sprintf(Life.Output[0], "%ld",  atol(Life.Output[0]) * atol(Life.Output[0]));
/*				Life.Output[1] = Life.Output[1] * Life.Output[1];
				Life.Output[2] = Life.Output[2] * Life.Output[2];
				//for (k = 0; k < Life.codelen-1; k++) // Learn from myself? other creature
				//Life.Code[Life.codelen+k] = Life.Code[k+1];
				//Life.codelen = Life.codelen+k;
				break;
			case 4: //Life.Child = true;
				Life.Output[0]--;
				Life.Output[1]--;
				Life.Output[2]--;
				break;
			case 5: Life.Output[0]++;
				Life.Output[1]++;
				Life.Output[2]++;
				break;
			case 6: Life.Output[0] = Life.Output[0] + Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] + Iteration->Input[1];
				Life.Output[2] = Life.Output[2] + Iteration->Input[2];
				break;
                        case 7: Life.Output[0] = Life.Output[0] - Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] - Iteration->Input[1];
				Life.Output[2] = Life.Output[2] - Iteration->Input[2];
				break;
                        case 8: Life.Output[0] = Life.Output[0] * Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] * Iteration->Input[1];
				Life.Output[2] = Life.Output[2] * Iteration->Input[2];
				break;
                        case 9: Life.Output[0] = Life.Output[0] / Iteration->Input[0]; 
				Life.Output[1] = Life.Output[1] / Iteration->Input[1];
				Life.Output[2] = Life.Output[2] / Iteration->Input[2];
				break;*/
		}
		Life.codepos++;
		if (Life.codepos > Life.codelen) Life.codepos = 0;
		}
		Life.TimeLeft--;
		Life.Energy--;
	}
                // PrintLife
//                printf("\n\rFunction:PrintLife Energy:%i Velocity:%i TimeLeft:%i codelen:%i codepos: %i parentref: %i ref: %i \nCode:",
//                Life.Energy, Life.Velocity, Life.TimeLeft, Life.codelen, Life.codepos, Life.ParentRef, Life.Ref);
//                for (int k = 0; k < Life.codelen; k++) printf("%i", Life.Code[k]);

		Iteration->Lifes[i] = Life;
	}
//        printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%i Energy:%i NumOfLifes:%i AliveCreatures: %i",
//        Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

void NewWorld(World *Iteration)
{
        Iteration->Energy = 0;
        Iteration->TimeLeft = 1500000;
        Iteration->NumOfLifes = 0;
        Iteration->MaxEnergy = 50;
        Iteration->AliveCreatures = 0;
	long I0 = atol(strcpy(Iteration->Input[0], "5"));
//	Iteration->Fitness = ((((Iteration->Input + Iteration->Input + 1) * Iteration->Input) - Iteration->Input) / Iteration->Input) + Iteration->Input - 1;
	// Code:9,9,4,6,9,5,7,9,5,4,5,3,4,6,3,8,5,
	sprintf(Iteration->Fitness[0], "%ld", (((I0 * I0) * I0 + 1 + I0) - 1) * I0);
        long I1 = atol(strcpy(Iteration->Input[1], "10"));
	sprintf(Iteration->Fitness[1], "%ld", (((I1 * I1) * I1 + 1 + I1) - 1) * I1);
        long I2 = atol(strcpy(Iteration->Input[2], "0"));
        sprintf(Iteration->Fitness[2], "%ld", (((I2 * I2) * I2 + 1 + I2) - 1) * I2);
	for (int i = 0; i < 2; i++)
	{
	        InitLife(Iteration, 0);
	}
	Creature ArtLife = InitLife(Iteration, -1);
//	ArtLife.Code = {5,1,8,2,6,6,1,3,3,1,6};
	ArtLife.Code[0] = 8;
	ArtLife.Code[1] = 4;
	ArtLife.codelen = 2;
}

void PrintWorld(World *Iteration)
{
	printf("\n\r------------------------\n\rFunction:PrintWorld TimeLeft:%ld Energy:%i NumOfLifes:%i AliveCreatures: %i\n--------------------", 
	Iteration->TimeLeft, Iteration->Energy, Iteration->NumOfLifes, Iteration->AliveCreatures);
}

int main(int argc, char **argv)
{
        time_t t;

        // Intializes random number generator
        srand((unsigned) time(&t));

	// set up device
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("device %d: %s \n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));	

	// allocate host memory
	int nElem = 1<22;
	size_t nBytes = sizeof(World);
	World     *h_A = (World *)malloc(nBytes);
	World *hostRef = (World *)malloc(nBytes);
	World *gpuRef  = (World *)malloc(nBytes);

	// initialize host array
	NewWorld(gpuRef);

	// allocate device memory
	World *d_A, *d_C;

	CHECK(hipMalloc((World**)&d_A, nBytes));
        CHECK(hipMalloc((World**)&d_C, nBytes));
	
	// copy data from host to device
//	CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));

        PrintLife(&gpuRef->Lifes[0]);
        PrintLife(&gpuRef->Lifes[1]);
        PrintLife(&h_A->Lifes[2]);

	PrintWorld(gpuRef);

        char BestFit[3][50];

	sprintf(BestFit[0], "%ld", abs(atol(gpuRef->Fitness[0]) - atol(gpuRef->Lifes[0].Output[0])));
	sprintf(BestFit[1], "%ld", abs(atol(gpuRef->Fitness[1]) - atol(gpuRef->Lifes[0].Output[1])));
	sprintf(BestFit[2], "%ld", abs(atol(gpuRef->Fitness[2]) - atol(gpuRef->Lifes[0].Output[2])));
	
	int BestFitNo = 0;

        // Run World all iterations
	do
        {

                for (int j = 0; j < gpuRef->NumOfLifes; j++) {
	                strcpy(gpuRef->Lifes[j].Output[0], gpuRef->Input[0]);
			strcpy(gpuRef->Lifes[j].Output[1], gpuRef->Input[1]);
			strcpy(gpuRef->Lifes[j].Output[2], gpuRef->Input[2]);
		}

                // copy data from host to device
                CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));

//		RunLife <<<1, gpuRef->NumOfLifes>>>(d_A, 1<<22);
	        RunLife <<<64, 512>>>(d_A, 1<<22);
		CHECK(hipDeviceSynchronize());
	        CHECK(hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost));
		gpuRef->AliveCreatures = 0;
		gpuRef->Energy = 0;
	        BestFitNo = gpuRef->NumOfLifes-1;

//		BestFit[0] = abs(gpuRef->Fitness[0] - gpuRef->Lifes[BestFitNo].Output[0]);
//		BestFit[1] = abs(gpuRef->Fitness[1] - gpuRef->Lifes[BestFitNo].Output[1]);
//	        BestFit[2] = abs(gpuRef->Fitness[2] - gpuRef->Lifes[BestFitNo].Output[2]);

	        sprintf(BestFit[0], "%ld", abs(atol(gpuRef->Fitness[0]) - atol(gpuRef->Lifes[0].Output[0])));
		sprintf(BestFit[1], "%ld", abs(atol(gpuRef->Fitness[1]) - atol(gpuRef->Lifes[0].Output[1])));
	        sprintf(BestFit[2], "%ld", abs(atol(gpuRef->Fitness[2]) - atol(gpuRef->Lifes[0].Output[2])));

		for (int j = 0; j < gpuRef->NumOfLifes; j++) {
//			PrintLife(&gpuRef->Lifes[j]);
//                        printf(">>%d", gpuRef->ChildLifes[j]);
			if (gpuRef->Lifes[j].Energy > 0 && gpuRef->Lifes[j].TimeLeft > 0) 
			{
//	                        PrintLife(&gpuRef->Lifes[j]);
				gpuRef->AliveCreatures++;
				gpuRef->Energy += gpuRef->Lifes[j].Energy;
//	                        PrintLife(&gpuRef->Lifes[j]);
//                                printf(" *** BestFit[0] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[0], gpuRef->Lifes[j].Output[0], abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]), BestFit[0]);
//                                printf(" *** BestFit[1] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[1], gpuRef->Lifes[j].Output[1], abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]), BestFit[1]);
//                                printf(" *** BestFit[2] = %ld - %ld = %ld vs CurBestFit %ld", gpuRef->Fitness[2], gpuRef->Lifes[j].Output[2], abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]), BestFit[2]);
//			if (abs(gpuRef->Fitness[0] - gpuRef->Lifes[j].Output[0]) < BestFit[0] && abs(gpuRef->Fitness[1] - gpuRef->Lifes[j].Output[1]) < BestFit[1] && abs(gpuRef->Fitness[2] - gpuRef->Lifes[j].Output[2]) < BestFit[2]) {
			if (abs(atol(gpuRef->Fitness[0]) - atol(gpuRef->Lifes[j].Output[0])) + abs(atol(gpuRef->Fitness[1]) - atol(gpuRef->Lifes[j].Output[1])) + abs(atol(gpuRef->Fitness[2]) - atol(gpuRef->Lifes[j].Output[2])) 
			< atol(BestFit[0]) + atol(BestFit[1]) + atol(BestFit[2])) {
				printf("\n *** BestFit vs NewBestFit : %ld# vs %ld#", atol(BestFit[0]) + atol(BestFit[1]) + atol(BestFit[2]), abs(atol(gpuRef->Fitness[0]) - atol(gpuRef->Lifes[j].Output[0])) + abs(atol(gpuRef->Fitness[1]) - atol(gpuRef->Lifes[j].Output[1])) + abs(atol(gpuRef->Fitness[2]) - atol(gpuRef->Lifes[j].Output[2])));
				sprintf(BestFit[0], "%ld", abs(atol(gpuRef->Fitness[0]) - atol(gpuRef->Lifes[j].Output[0])));
                                sprintf(BestFit[1], "%ld", abs(atol(gpuRef->Fitness[1]) - atol(gpuRef->Lifes[j].Output[1])));
                                sprintf(BestFit[2], "%ld", abs(atol(gpuRef->Fitness[2]) - atol(gpuRef->Lifes[j].Output[2])));
				BestFitNo = j;
//				printf(" *** BestFit[0] = %ld - %ld = %ld", gpuRef->Lifes[j].Output[0], gpuRef->Fitness[0], BestFit[0]);
				if (atol(BestFit[0]) == 0 && atol(BestFit[1]) == 0 && atol(BestFit[2]) == 0) {
					PrintLife(&gpuRef->Lifes[j]);
	                                printf(" *** BestFit[0] = %s - %s = %ld vs CurBestFit %s", gpuRef->Fitness[0], gpuRef->Lifes[j].Output[0], abs(atol(gpuRef->Fitness[0]) - atol(gpuRef->Lifes[j].Output[0])), BestFit[0]);
		                        printf(" *** BestFit[1] = %s - %s = %ld vs CurBestFit %s", gpuRef->Fitness[1], gpuRef->Lifes[j].Output[1], abs(atol(gpuRef->Fitness[1]) - atol(gpuRef->Lifes[j].Output[1])), BestFit[1]);
			                printf(" *** BestFit[2] = %s - %s = %ld vs CurBestFit %s", gpuRef->Fitness[2], gpuRef->Lifes[j].Output[2], abs(atol(gpuRef->Fitness[2]) - atol(gpuRef->Lifes[j].Output[2])), BestFit[2]);
					break;
				}
			}
			}
		}
		int p = 0;
		for (int n = 0; n < range_rand(10, 30); n++) 
		{
			for (p = p; p < gpuRef->NumOfLifes; p++) if (gpuRef->Lifes[p].TimeLeft <= 0 || gpuRef->Lifes[p].Energy <= 0) break;
			printf("\n ** Slot for new life is %i", p);
                        PrintLife(&gpuRef->Lifes[p]);
			gpuRef->Lifes[p].Energy = 29;
                        gpuRef->Lifes[p].TimeLeft = 29;
                        gpuRef->Lifes[p].Velocity = 1;
			if (range_rand(1, 4) == 1) {
				gpuRef->Lifes[p].codelen = gpuRef->Lifes[BestFitNo].codelen / 2;
			} else if (range_rand(1, 4) == 1) {
				gpuRef->Lifes[p].codelen = gpuRef->Lifes[BestFitNo].codelen * 2;
				if (gpuRef->Lifes[p].codelen > 49) gpuRef->Lifes[p].codelen = 49;
			} else {
				gpuRef->Lifes[p].codelen = gpuRef->Lifes[BestFitNo].codelen;
			}
                        gpuRef->Lifes[p].codepos = 0;
		        for (int k = 0; k < gpuRef->Lifes[BestFitNo].codelen; k++) {
				if (range_rand(1, 2) == 1) {
	                                gpuRef->Lifes[p].Code[k] = range_rand(1, 9);		
				}
				else { 
					gpuRef->Lifes[p].Code[k] = gpuRef->Lifes[BestFitNo].Code[k];
				}
			}
			gpuRef->Lifes[p].Ref = p;
                        gpuRef->Lifes[p].ParentRef = gpuRef->Lifes[BestFitNo].Ref;
			strcpy(gpuRef->Lifes[p].Output[0], "0");
                        strcpy(gpuRef->Lifes[p].Output[1], "0");
                        strcpy(gpuRef->Lifes[p].Output[2], "0");
                        PrintLife(&gpuRef->Lifes[BestFitNo]);
//			printf(" %ld#%ld#%ld#%ld", BestFit[0], BestFit[1], BestFit[2], BestFit[0] + BestFit[1] + BestFit[2]);
//                        printf("\n %ld#", BestFit[0] + BestFit[1] + BestFit[2]);

//				printf("\n *** Parent: %i", j);
//                        printf("\n ***LIFE IS BORN from %i", gpuRef->Lifes[BestFitNo].Ref);
                        PrintLife(&gpuRef->Lifes[p]);
                        if (p >= gpuRef->NumOfLifes) gpuRef->NumOfLifes++;
		}
                PrintWorld(gpuRef);
		// copy data from host to device
//	        CHECK(hipMemcpy(d_A, gpuRef, nBytes, hipMemcpyHostToDevice));
		if (BestFit[0] == 0 && BestFit[1] == 0 && BestFit[2] == 0) break;
	} while (gpuRef->Energy > 0 && gpuRef->TimeLeft > 0);

	CHECK(hipDeviceSynchronize());
	CHECK(hipMemcpy(gpuRef, d_A, nBytes, hipMemcpyDeviceToHost));

	PrintWorld(gpuRef);

	CHECK(hipGetLastError());;

	printf("\n\n ### THE WINNER IS %i", BestFitNo);
	PrintLife(&gpuRef->Lifes[BestFitNo]);

/*	printf("\n\n *** Admire the winners genomes history:");
        for (int i = 0; i < gpuRef->NumOfLifes; i++)
	{
		Creature Parent = gpuRef->Lifes[i];
		if (IsAlive(&Parent)) {
			PrintLife(&Parent);
			while (Parent.ParentRef > 0) {
				Parent = FindCreature(gpuRef, Parent.ParentRef);
				printf("->");
				PrintCode(&Parent);
//				PrintLife <<<1,1>>>(Parent);
			}
		}
	}*/
/*        printf("\n\n *** Admire the winners story:");
        for (int i = 0; i < gpuRef->NumOfLifes; i++)
        {
                Creature Parent = gpuRef->Lifes[i];
                if (IsAlive(&Parent)) {
//                        PrintLife(Parent);
			printf("\n");
                        while (Parent.ParentRef > 0) {
				printf("%i->", Parent.Ref);
                                Parent = FindCreature(gpuRef, Parent.ParentRef);
//                              PrintLife <<<1, 1>>>(Parent);
                        }
                }
        }
*/
	printf("\n");
}
